#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "timer.hpp"

__host__ inline void checkCudaError(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        printf("CUDA error: %s %s %d\n", hipGetErrorString(code), file, line);
    }
}

#define CHECKCUDAERR(ans)                          \
    {                                              \
        checkCudaError((ans), __FILE__, __LINE__); \
    }


__global__ void addOneToEachElement(uint64_t *data, uint64_t N) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        data[idx] = __brev(data[idx]);
    }
}

void process(uint64_t *data, uint64_t N) {
    uint64_t *d_data;
    hipMalloc(&d_data, N * sizeof(uint64_t));

    hipMemcpy(d_data, data, N * sizeof(uint64_t), hipMemcpyHostToDevice);

    int blockSize = 64;
    int numBlocks = (N + blockSize - 1) / blockSize;
    addOneToEachElement<<<numBlocks, blockSize>>>(d_data, N);
    CHECKCUDAERR(hipGetLastError());

    hipMemcpy(data, d_data, N * sizeof(uint64_t), hipMemcpyDeviceToHost);
    for (uint64_t i = 0; i < 8; ++i) {
        printf("h_data: %lu\n", data[i]);
    }

    hipFree(d_data);
}

int main() {
    int deviceCount;
    CHECKCUDAERR(hipGetDeviceCount(&deviceCount));
    printf("Detected %d CUDA Capable device(s).\n", deviceCount);

    hipStream_t ss;
    hipSetDevice(0);
    CHECKCUDAERR(hipStreamCreate(&ss));

    uint64_t N = (1<<31);
    uint64_t *data;
    hipMallocManaged(&data, N * sizeof(uint64_t), hipMemAttachHost);


    // 初始化host memory数据
    for (uint64_t i = 0; i < N; ++i) {
        data[i] = i;
    }

    for (uint64_t i = 0; i < 8; ++i) {
        printf("data: %lu\n", data[i]);
    }

    uint64_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);
    printf("1. free_mem: %lu, total_mem: %lu\n", free_mem>>20, total_mem>>20);

    int blockSize = 64;
    int numBlocks = (N + blockSize - 1) / blockSize;
    addOneToEachElement<<<numBlocks, blockSize, 0, ss>>>(data, N);
    CHECKCUDAERR(hipGetLastError());
    CHECKCUDAERR(hipStreamSynchronize(ss));

    for (uint64_t i = 0; i < 8; ++i) {
        printf("data: %lu\n", data[i]);
    }

    hipMemGetInfo(&free_mem, &total_mem);
    printf("2. free_mem: %lu, total_mem: %lu\n", free_mem>>20, total_mem>>20);

    CHECKCUDAERR(hipMemPrefetchAsync(data, N * sizeof(uint64_t), hipCpuDeviceId, ss));
    CHECKCUDAERR(hipStreamSynchronize(ss));

    hipMemGetInfo(&free_mem, &total_mem);
    printf("3. free_mem: %lu, total_mem: %lu\n", free_mem>>20, total_mem>>20);

    uint64_t *h_data = (uint64_t *)malloc(N * sizeof(uint64_t));
    memcpy(h_data, data, N * sizeof(uint64_t));
    TimerStart(malloc_test);
    process(h_data, N);
    TimerStopAndLog(malloc_test);
    free(h_data);

    uint64_t *h_data1 = (uint64_t *)hipHostMalloc(N * sizeof(uint64_t));
    memcpy(h_data1, data, N * sizeof(uint64_t));
    TimerStart(cudaMallocHost_test);
    process(h_data1, N);
    TimerStopAndLog(cudaMallocHost_test);
    hipHostFree(h_data1);

    uint64_t *h_data2 = (uint64_t *)hipMallocManaged(N * sizeof(uint64_t));
    memcpy(h_data2, data, N * sizeof(uint64_t));
    TimerStart(cudaMallocManaged_test);
    process(h_data2, N);
    TimerStopAndLog(cudaMallocManaged_test);
    hipHostFree(h_data2);

    hipFree(data);

    return 0;
}
